#include "hip/hip_runtime.h"
/* Program for finding out majority element in an array */
#include<stdio.h>
#include <iostream>
#include <cmath>
#include <vector>
#include "timer.h"

#define bool int
#define EPS 0.000001
#define MAX_LENGTH_BLOCK 40
// MAX_DATA_BLOCK_ON_THREAD_BLOCK
#define MAX_DB_ON_TB 32

inline void cudaCheck(const hipError_t &err, const std::string &mes) {
	if (err != hipSuccess) {
		std::cout << (mes + " - " + hipGetErrorString(err)) << std::endl;
		exit(EXIT_FAILURE);
	}
}

__device__ __host__ bool floatEquals(float a, float b) {
    if (fabs(a - b) < EPS) {
        return true;
    } else {
        return false;
    }
}

/* Function to find the candidate for Majority */
float findCandidate(const float * const a, int size) {
    int maj_index = 0, count = 1;
    int i;
    for(i = 1; i < size; i++) {
        if(floatEquals(a[maj_index], a[i])) {
            count++;
        } else {
            count--;
        }
        if(count == 0) {
            maj_index = i;
            count = 1;
        }
    }
    return a[maj_index];
}

/* Function to check if the candidate occurs more than n/2 times */
bool isMajority(const float * const a, int size, float cand) {
    int i, count = 0;
    for (i = 0; i < size; i++) {
        if(floatEquals(a[i], cand)) {
            count++;
        }
    }
    if (count > size/2) {
       return true;
    } else {
       return false;
    }
}

/* Function to print Majority Element */
int findMajority(const float * const a, int size) {
    /* Find the candidate for Majority*/
    float cand = findCandidate(a, size);

    if(isMajority(a, size, cand)) {
        return cand;
    } else {
        return -1;
    }
}

// This kernel is optimized for the maximum dimension
// of the input data is 100 count of blocks and 60 length of block.
__global__ void findMajorityKernel(
    const float * const data,
    int *results,
    const int length,
	const int countBlocks
) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ float dataOfBlockShared[MAX_LENGTH_BLOCK * MAX_DB_ON_TB];
	for (int i = 0; i < length; i += 1) {
		dataOfBlockShared[i * blockDim.x + threadIdx.x] = data[blockIdx.x * MAX_DB_ON_TB + i * countBlocks + threadIdx.x];
	}
	__syncthreads();
	const float * const dataOfBlock = &dataOfBlockShared[threadIdx.x];
	if (index < countBlocks) {
        int maj_index = 0;
        int count = 1;
        for(int i = 1; i < length; i++) {
            if(floatEquals(dataOfBlock[maj_index * MAX_DB_ON_TB], dataOfBlock[i * MAX_DB_ON_TB])) {
                count++;
            } else {
                count--;
            }
            if(count == 0) {
                maj_index = i;
                count = 1;
            }
        }
        int cand = dataOfBlock[maj_index * MAX_DB_ON_TB];

        int countCheck = 0;
        for (int i = 0; i < length; i++) {
            if(floatEquals(dataOfBlock[i * MAX_DB_ON_TB], cand)) {
                countCheck++;
            }
        }
		int result = -1;
        if (countCheck > length / 2) {
           result = cand;
	    }
		results[index] = result;
    }
}

/* Driver function to test above functions */
int main(int argc, char * argv[]) {
	try {
	    int length = 10;
	    int countBlocks = 1;
	    int countIter = 1000;
	    if  (argc > 1) {
	        length = atoi(argv[1]);
	    }
	    if (argc > 2) {
	        countBlocks = atoi(argv[2]);
	    }
	    const int sizeOfData = length * countBlocks;
		const int sizeOfDataBytes = length * countBlocks * sizeof(float);
	    std::vector<float> data(sizeOfData);
	    for (int i = 0; i < sizeOfData; ++i) {
	        // data[i] = rand() % (length / 3);
			data[i] = rand() % 2;
	        // std::cout << data[i] << " ";
	        // if (i % length == (length - 1)) {
	        //     std::cout << std::endl;
	        // }
	    }
	    // std::cout << std::endl;

	    // cuda implementation
	    Timer timer;
	    hipError_t err = hipSuccess;
	    float *dataDev;
	    err = hipMalloc((void **)&dataDev, sizeOfDataBytes);
	    cudaCheck(err, "failed to allocated dataDev");
	    int *resultGPUDev;
	    err = hipMalloc((void **)&resultGPUDev, countBlocks * sizeof(int));
	    cudaCheck(err, "failed to allocated resultGPUDev");

		std::vector<int> resultGPU(countBlocks);
	    float computeTimeGPU = 0.0f;
	    float computeTimeWithCopyGPU = 0.0f;
	    for (int i = 0; i < countIter; ++i) {
	        timer.begin("with copy");
			timer.begin("change data");
			std::vector<float> data1(sizeOfData);
			for (int j = 0; j < countBlocks; ++j) {
				for (int k = 0; k < length; ++k) {
					data1[k * countBlocks + j] = data[j * length + k];
				}
			}
			timer.end("change data");
			err = hipMemcpy(dataDev, &data1[0], sizeOfDataBytes, hipMemcpyHostToDevice);
			cudaCheck(err, "failed to copy data to the GPU");

	        int threadsPerBlock = 32;
			int blocksPerGrid = (countBlocks + threadsPerBlock - 1) / threadsPerBlock;
			timer.begin("compute");
			findMajorityKernel<<<blocksPerGrid, threadsPerBlock>>>(dataDev, resultGPUDev, length, countBlocks);
			hipDeviceSynchronize();
			timer.end("compute");
			err = hipGetLastError();
			cudaCheck(err, "failed to launch kernel");

			err = hipMemcpy(&resultGPU[0], resultGPUDev, countBlocks * sizeof(int), hipMemcpyDeviceToHost);
			cudaCheck(err, "failed to copy resultGPUDev to host");
			timer.end("with copy");
			computeTimeGPU += timer.getTimeMillisecondsFloat("compute") + timer.getTimeMillisecondsFloat("change data");
			computeTimeWithCopyGPU += timer.getTimeMillisecondsFloat("with copy");
	    }

		int countOfSucccesSearch = 0;
		float computeTimeCPU = 0.0f;
	    std::vector<int> resultCPU(countBlocks);
		for (int j = 0; j < countIter; ++j) {
		    for (int i = 0; i < countBlocks; ++i) {
				timer.begin("compute");
		        int result = findMajority(&data[length * i], length);
					resultCPU[i] = result;
				if (result != -1 && j == 0) {
					countOfSucccesSearch++;
				}
				timer.end("compute");
				computeTimeCPU += timer.getTimeMillisecondsFloat("compute");
		    }
		}
		std::cout << "count of success search " << countOfSucccesSearch << std::endl;
	    // for (auto &result : resultCPU) {
	    //     std::cout << result << " ";
	    // }
		// std::cout << std::endl;
		// for (auto &result : resultGPU) {
	    //     std::cout << result << " ";
	    // }
	    // std::cout << std::endl;

		// check correct work
		for (int i = 0; i < countBlocks; ++i) {
			if (resultCPU[i] != resultGPU[i]) {
				std::cout << "gpu and cpu results isn't equal." << std::endl;
				break;
			}
		}

		const int countOperations = countBlocks * log2f(length);
		const float avgComputeTimeGPU = computeTimeGPU / countIter;
		const float avgComputeTimeWithCopyGPU = computeTimeWithCopyGPU / countIter;
		std::cout << "avg compute time GPU = " << avgComputeTimeGPU << " milliseconds" << std::endl;
		std::cout << "avg compute time(with copy) GPU = " << avgComputeTimeWithCopyGPU << " milliseconds" << std::endl;
		std::cout << "Computational throughput GPU = " << countOperations / (avgComputeTimeGPU * 10e3) << " B/s" << std::endl;
		std::cout << "Computational throughput(with copy) GPU = " << countOperations / (avgComputeTimeWithCopyGPU * 10e3) << " B/s" << std::endl;

		const float avgComputeTimeCPU = computeTimeCPU / countIter;
		std::cout << "avg compute time CPU = " << avgComputeTimeCPU << " milliseconds" << std::endl;
		std::cout << "Computational throughput CPU = " << countOperations / (avgComputeTimeCPU * 10e3) << " B/s" << std::endl;
	} catch (const std::string &mes) {
		std::cout << "An error is occured - " << mes << std::endl;
	}

    // getchar();
    return 0;
}
